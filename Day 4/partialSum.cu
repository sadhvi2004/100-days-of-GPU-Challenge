
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void partialSumKernel(int *input, int *output, int n) {
    // Shared memory 
    extern __shared__ int sharedMemory[];
    
    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x*2 + tid;

    if (index < n) {
        // Load input into shared memory and optimize the loading to do coalescing 
        sharedMemory[tid] = input[index]+input[index+blockDim.x];
        __syncthreads();

        // Perform inclusive scan in shared memory
        for (int stride = 1; stride < blockDim.x; stride *= 2) {
            int temp = 0;
            if (tid >= stride) {
                temp = sharedMemory[tid - stride];
            }
            __syncthreads();
            sharedMemory[tid] += temp;
            __syncthreads();
        }

        // Write result to global memory
        output[index] = sharedMemory[tid];
    }
}

int main() {
    const int N = 16;
    const int blockSize = 8;

    int h_input[N] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16};
    int h_output[N];

    int *d_input, *d_output;
    size_t size = N * sizeof(int);

    hipMalloc(&d_input, size);
    hipMalloc(&d_output, size);

    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

    partialSumKernel<<<N / blockSize, blockSize, blockSize * sizeof(int)>>>(d_input, d_output, N);

    hipMemcpy(h_output, d_output, size, hipMemcpyDeviceToHost);

 
    printf("Input: ");
    for (int i = 0; i < N; i++) {
        printf("%d ", h_input[i]);
    }
    printf("\nOutput: ");
    for (int i = 0; i < N; i++) {
        printf("%d ", h_output[i]);
    }
    printf("\n");

    hipFree(d_input);
    hipFree(d_output);

    return 0;
}