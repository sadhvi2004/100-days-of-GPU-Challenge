
#define LOAD_SIZE 32
#include <iostream>
#include <hip/hip_runtime.h>
// going to code Brent-Kung algorithm
__global__ void prefixsum_kernel(float* A,float* C,int N){
  int threadId=threadIdx.x;
  int i=2*blockDim.x*blockIdx.x+threadId;

  //load in shared memory

  __shared__ float S_A[LOAD_SIZE];
  if (i<N){
    S_A[threadId]=A[i];
  }
  if (i+blockDim.x<N){
    S_A[threadId+blockDim.x]=A[i+blockDim.x];
  }
  __syncthreads();

for(int jump=1;jump<=blockDim.x;jump*=2){
  //I need to sync the threads because I need all their values for the next iteration
  __syncthreads();
  int j= jump*2*(threadId+1) -1;
  if (j<LOAD_SIZE){
    //I think this will make the threads in the warp inactive, but just a first approximation I'm going to do it like this.

    S_A[j]+=S_A[j-jump];
  }}
  __syncthreads();

//Now the reduction part
//just by pattern recognition the tree is flipped so I assume we just flip the previous algorithm somehow.


for(int jump=LOAD_SIZE/4;jump>=1;jump/=2){
  //I need to sync the threads because I need all their values for the next iteration
  __syncthreads();
  int j= jump*2*(threadId+1) -1;
  if (j<LOAD_SIZE-jump){

     S_A[j+jump]+=S_A[j];
  }
  __syncthreads();
}
if (i<N) C[i]=S_A[threadId];
if (i<N-blockDim.x) C[i+blockDim.x]=S_A[threadId+blockDim.x];
__syncthreads();
  

}


void checkCudaError(const char *message) {
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error (%s): %s\n", message, hipGetErrorString(error));
        exit(-1);
    }
}


int main(){
  int N=10;
  float A[N],C[N];
for (int i = 0; i < N; i++) {
    A[i] = i + 1.0f;
}
  float* d_A;
  float* d_C;
  hipMalloc(&d_A,N*sizeof(float));
  hipMalloc(&d_C,N*sizeof(float));
  hipMemcpy(d_A,A,N*sizeof(float),hipMemcpyHostToDevice);
  checkCudaError("Failed to copy input data to device");
  dim3 dimBlock(32);
  dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x);
  prefixsum_kernel<<<dimGrid, dimBlock>>>(d_A,d_C,N);
  checkCudaError("Failed to execute the kernel");
  hipDeviceSynchronize();
  hipMemcpy(C,d_C,N*sizeof(float),hipMemcpyDeviceToHost);
checkCudaError("Failed to copy output data to host");

hipFree(d_A);
hipFree(d_C);


//printing the results
printf("A:\n");
for (int i=0; i<N;i++){
  printf("%.2f ", A[i]);

}
printf("C:\n");
for (int i=0; i<N;i++){
  printf("%.2f ", C[i]);

}
}
